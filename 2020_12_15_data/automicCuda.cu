#include <iostream>

#include "hip/hip_runtime.h"
#include "time.h"

using namespace std;

#define num (256 * 1024 * 1024)


__global__ void hist(unsigned char* inputdata, int* outPutHist, long size) {
 
  __shared__ int temp[256];
  temp[threadIdx.x] = 0;
  __syncthreads();

  // �����߳��������߳�ƫ����
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  while (ids < size) {
    //����ԭ�Ӳ�����һ��block�е����ݽ���ֱ��ͼͳ��
    atomicAdd(&temp[inputdata[ids]], 1);
    ids += offset;
  }

  // �ȴ�ͳ����ɣ���ȥͳ�ƽ��
  __syncthreads();
  atomicSub(&outPutHist[threadIdx.x], temp[threadIdx.x]);
}

int main() {
  // ����������� [0 255]
  unsigned char* cpudata = new unsigned char[num];
  for (size_t i = 0; i < num; i++)
    cpudata[i] = static_cast<unsigned char>(rand() % 256);

  // �����������ڼ�¼ͳ�ƽ��
  int cpuhist[256];
  memset(cpuhist, 0, 256 * sizeof(int));

  /*******************************   CPU���Դ���
   * *********************************/
  clock_t cpu_start, cpu_stop;
  cpu_start = clock();
  for (size_t i = 0; i < num; i++) cpuhist[cpudata[i]]++;
  cpu_stop = clock();
  cout << "CPU time: " << (cpu_stop - cpu_start) << "ms" << endl;

  /*******************************   GPU���Դ���
   * *********************************/

  //�����¼����ڼ�ʱ
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //�����Դ沢������copy���Դ�
  unsigned char* gpudata;
  hipMalloc((void**)&gpudata, num * sizeof(unsigned char));
  hipMemcpy(gpudata, cpudata, num * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  // �����Դ����ڴ洢�������,����CPU�ļ�����copy��ȥ
  int* gpuhist;
  hipMalloc((void**)&gpuhist, 256 * sizeof(int));
  hipMemcpy(gpuhist, cpuhist, 256 * sizeof(int), hipMemcpyHostToDevice);

  // ִ�к˺�������ʱ
  hipEventRecord(start, 0);
  hist<<<1024, 256>>>(gpudata, gpuhist, num);
  hipEventRecord(stop, 0);

  // �����copy������
  int histcpu[256];
  hipMemcpy(cpuhist, gpuhist, 256 * sizeof(int), hipMemcpyDeviceToHost);

  // ���ٿ��ٵ��ڴ�
  hipFree(gpudata);
  hipFree(gpuhist);
  delete cpudata;

  // ����GPU����ʱ�䲢���ټ�ʱ�¼�
  hipEventSynchronize(stop);
  float gputime;
  hipEventElapsedTime(&gputime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  cout << "GPU time: " << gputime << "ms" << endl;

  // ��֤���
  long result = 0;
  for (size_t i = 0; i < 256; i++) result += cpuhist[i];
  if (result == 0)
    cout << "GPU has the same result with CPU." << endl;
  else
    cout << "Error: GPU has a different result with CPU." << endl;

  system("pause");
  return 0;
}