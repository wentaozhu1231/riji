#include "hip/hip_runtime.h"
#include <stdlib.h>

#include <iostream>

#include "hip/hip_runtime.h"
#include ""

using namespace std;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();

  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

#define n 10
texture<float, 1, hipReadModeElementType> tex_a;
texture<float, 1, hipReadModeElementType> tex_b;

__global__ void test_texture(float *dev_test_a, float *dev_test_b,
                             float *dev_result_c) {
  int offset = threadIdx.x + blockIdx.x * blockDim.x;
  float a = tex1Dfetch(tex_a, offset);
  float b = tex1Dfetch(tex_b, offset);
  dev_result_c[offset] = a + b;
}

int main() {
  float *test_a = NULL, *test_b = NULL, *result_c = NULL;

  float *dev_test_a = NULL, *dev_test_b = NULL, *dev_result_c = NULL;

  hipHostAlloc((void **)&test_a, n * sizeof(float), hipHostMallocMapped);
  hipHostAlloc((void **)&test_b, n * sizeof(float), hipHostMallocMapped);
  hipHostAlloc((void **)&result_c, n * sizeof(float), hipHostMallocMapped);

  for (int i = 0; i < n; i++) {
    test_a[i] = i;
    test_b[i] = i * i;
  }
  hipMalloc((void **)&dev_test_a, n * sizeof(float));
  hipMalloc((void **)&dev_test_b, n * sizeof(float));
  hipMalloc((void **)&dev_result_c, n * sizeof(float));

  hipError_t cudaStatus;

  cudaStatus =
      hipMemcpy(dev_test_a, test_a, n * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    cout << "hipMemcpy dev_test_a failed!" << endl;
    exit(EXIT_FAILURE);
  }

  cudaStatus =
      hipMemcpy(dev_test_b, test_b, n * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    cout << "hipMemcpy dev_test_b failed!" << endl;
    exit(EXIT_FAILURE);
  }

  hipBindTexture(0, &tex_a, dev_test_a, &tex_a.channelDesc, n * sizeof(float));
  checkCUDAError("binding  dex_test_a");
  hipBindTexture(0, &tex_b, dev_test_b, &tex_b.channelDesc, n * sizeof(float));
  checkCUDAError("binding dex_test_b");

  test_texture<<<5, 10>>>(dev_test_a, dev_test_b, dev_result_c);

  cudaStatus = hipMemcpy(result_c, dev_result_c, n * sizeof(float),
                          hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    cout << "hipMemcpy result_c failed!" << endl;
    exit(EXIT_FAILURE);
  }

  cout << "Texture test result:" << endl;

  for (int i = 0; i < n; i++) {
    cout << test_a[i] << " " << test_b[i] << " " << result_c[i] << endl;
  }

  hipUnbindTexture(tex_a);
  hipUnbindTexture(tex_b);

  hipHostFree(test_a);
  hipHostFree(test_b);
  hipHostFree(result_c);
  hipFree(dev_test_a);
  hipFree(dev_test_b);
  hipFree(dev_result_c);

  cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceReset failed!/n");
    return EXIT_FAILURE;
  }

  return 0;
}